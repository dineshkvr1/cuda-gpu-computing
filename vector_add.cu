#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

// CUDA kernel for vector addition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// Function to check CUDA errors
void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << message << " - " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

int main() {
    // Vector size
    const int N = 1024;
    const size_t bytes = N * sizeof(float);
    
    // Host vectors
    std::vector<float> h_a(N), h_b(N), h_c(N);
    
    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }
    
    // Device vectors
    float *d_a, *d_b, *d_c;
    
    // Allocate device memory
    checkCudaError(hipMalloc(&d_a, bytes), "Failed to allocate device memory for vector a");
    checkCudaError(hipMalloc(&d_b, bytes), "Failed to allocate device memory for vector b");
    checkCudaError(hipMalloc(&d_c, bytes), "Failed to allocate device memory for vector c");
    
    // Copy data from host to device
    checkCudaError(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice), "Failed to copy vector a to device");
    checkCudaError(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice), "Failed to copy vector b to device");
    
    // Launch kernel
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    
    std::cout << "Launching CUDA kernel with " << gridSize << " blocks and " << blockSize << " threads per block" << std::endl;
    
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
    
    // Check for kernel launch errors
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    
    // Wait for kernel to complete
    checkCudaError(hipDeviceSynchronize(), "Device synchronization failed");
    
    // Copy result back to host
    checkCudaError(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost), "Failed to copy result to host");
    
    // Verify results
    bool success = true;
    for (int i = 0; i < N; i++) {
        float expected = h_a[i] + h_b[i];
        if (abs(h_c[i] - expected) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            success = false;
            break;
        }
    }
    
    if (success) {
        std::cout << "Vector addition completed successfully!" << std::endl;
        std::cout << "First 10 results:" << std::endl;
        for (int i = 0; i < 10; i++) {
            std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
        }
    }
    
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}
